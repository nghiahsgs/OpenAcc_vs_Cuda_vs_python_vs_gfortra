
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include<math.h>
void load_matrix_from_file(FILE * file, int nb_rows,int nb_cols, double* mat){
        for(int i = 0; i < nb_rows; i++){
                for(int j = 0; j < nb_cols; j++){
                        //Use lf format specifier, %c is for character
                        //if (!fscanf(file, "%lf", &mat[i][j])){ 
                        int index=i*nb_cols + j;
                        if (!fscanf(file, "%lf", &mat[index])){
                                break;
                        }
                }
        }
}
void load_matrix_2D_from_file(FILE * file, int nb_rows,int nb_cols, double* matrix){
        for(int i = 0; i < nb_rows; i++){
                for(int j = 0; j < nb_cols; j++){
                        int index=i*nb_cols + j;
                        if (!fscanf(file, "%lf", &matrix[index])){
                                break;
                        }
                }
        }
}
void load_matrix_1D_from_file(FILE * file, int n, double* matrix){
                for(int j = 0; j < n; j++){
                        if (!fscanf(file, "%lf", &matrix[j])){
                                break;
                        }
                }
}
__global__ void kernel_gpu(int N, double *A, double *B,  double *result){
        int r = blockIdx.x*blockDim.x+ threadIdx.x;
        int s = blockIdx.y*blockDim.y+ threadIdx.y;

        if(r < N && s<N){
                atomicAdd(result, A[r*N+s]+B[r*N+s]);
        }
}
int main(){
        int N=1000;
        double *h_A, *h_B, *h_result;
        double *d_A, *d_B, *d_result;

        //khai bao vung nho trong host
        h_A = (double*)malloc(N*N*sizeof(double));
        h_B = (double*)malloc(N*N*sizeof(double));
        h_result = (double*)malloc(sizeof(double));

        //khai bao vung nho trong device
        hipMalloc(&d_A, N*N*sizeof(double));
        hipMalloc(&d_B, N*N*sizeof(double));
        hipMalloc(&d_result,sizeof(double));
        //load matrix from file
        FILE *file;
        file=fopen("matrix_1000_1000_001.txt", "r");
        load_matrix_2D_from_file(file, N, N,h_A);
        fclose(file);

        FILE *file2;
        file2=fopen("matrix_1000_1000_002.txt", "r");
        load_matrix_2D_from_file(file2, N,N,h_B);
        fclose(file2);

        //printf("gia tri A[500][500] la %f",h_A[500*100+500]);

        //====start log time
        clock_t begin=clock();

        hipMemcpy(d_A, h_A, N*N*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, N*N*sizeof(double), hipMemcpyHostToDevice);

        //execute kernel
        int nb_hyper=8;
        dim3 nb_block(int(N/nb_hyper)+1,int(N/nb_hyper)+1,1);
        dim3 nb_thread_per_block(nb_hyper, nb_hyper,1);
        kernel_gpu<<<nb_block, nb_thread_per_block>>>(N, d_A, d_B, d_result);

        //copy result from device to host
        hipMemcpy(h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

        //====end log time
        clock_t end = clock();
        double time_spent=(double) (end-begin)/CLOCKS_PER_SEC;
        
        printf("Total time:%f\n",time_spent);
        printf("result :%f\n",h_result[0]/(N*N));

        return 0;
}

