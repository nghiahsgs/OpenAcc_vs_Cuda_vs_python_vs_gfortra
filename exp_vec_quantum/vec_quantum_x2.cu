
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include<math.h>
void load_matrix_2D_from_file(FILE * file, int nb_rows,int nb_cols, double* matrix){
        for(int i = 0; i < nb_rows; i++){
                for(int j = 0; j < nb_cols; j++){
                        int index=i*nb_cols + j;
                        if (!fscanf(file, "%lf", &matrix[index])){
                                break;
                        }
                }
        }
}
void load_matrix_1D_from_file(FILE * file, int n, double* matrix){
                for(int j = 0; j < n; j++){
                        if (!fscanf(file, "%lf", &matrix[j])){
                                break;
                        }
                }
}
__global__ void kernel_gpu(int N, double *C, double *B, double *E,  double *result){
        int r = blockIdx.x*blockDim.x+ threadIdx.x;
        int s = blockIdx.y*blockDim.y+ threadIdx.y;
        int q = blockIdx.z*blockDim.z+ threadIdx.z;

        if(r < N && s<N && q<N){
                atomicAdd(result, C[r*N+s]*B[s*N+q]*(cos(E[q]-E[r])-cos(E[s]-E[q]))/(10+E[s]-E[q]/2-E[r]/2));
        }
}
int main(){
        int N=2000;
        double *h_C, *h_B,*h_E, *h_result;
        double *d_C, *d_B,*d_E, *d_result;

        //====start log time
        clock_t begin=clock();

        //khai bao vung nho trong host
        h_C = (double*)malloc(N*N*sizeof(double));
        h_B = (double*)malloc(N*N*sizeof(double));
        h_E = (double*)malloc(N*sizeof(double));
        h_result = (double*)malloc(sizeof(double));

        //khai bao vung nho trong device
        hipMalloc(&d_C, N*N*sizeof(double));
        hipMalloc(&d_B, N*N*sizeof(double));
        hipMalloc(&d_E, N*sizeof(double));
        hipMalloc(&d_result,sizeof(double));

        //load matrix from file
        FILE *file;
        file=fopen("matrix_2000_2000_001.txt", "r");
        load_matrix_2D_from_file(file, N, N,h_C);
        fclose(file);

        FILE *file2;
        file2=fopen("matrix_2000_2000_002.txt", "r");
        load_matrix_2D_from_file(file2, N,N,h_B);
        fclose(file2);

        FILE *file3;
        file3=fopen("matrix_2000_1_001.txt", "r");
        load_matrix_1D_from_file(file3, N,h_E);
        fclose(file3);

        //printf("h_C[500][500] : %f \n",h_C[500*1000+500]);
        //printf("h_B[500][500] : %f \n",h_B[500*1000+500]);
        //printf("h_E[500] : %f\n",h_E[999]);
        hipMemcpy(d_C, h_C, N*N*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, N*N*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_E, h_E, N*sizeof(double), hipMemcpyHostToDevice);

        //execute kernel
        int nb_hyper=8;
        dim3 nb_block(int(N/nb_hyper)+1,int(N/nb_hyper)+1,int(N/nb_hyper)+1);
        dim3 nb_thread_per_block(nb_hyper, nb_hyper,nb_hyper);
        kernel_gpu<<<nb_block, nb_thread_per_block>>>(N, d_C, d_B,d_E, d_result);
 	
	//copy result from device to host
        hipMemcpy(h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

        //====end log time
        clock_t end = clock();
        double time_spent=(double) (end-begin)/CLOCKS_PER_SEC;

        printf("Total time:%f\n",time_spent);
        printf("result :%f\n",h_result[0]);

        return 0;
}
